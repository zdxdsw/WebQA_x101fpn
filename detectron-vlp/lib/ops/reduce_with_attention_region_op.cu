#include "hip/hip_runtime.h"
#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/math.h"
#include "reduce_with_attention_region_op.h"

#include <stdio.h>

namespace caffe2 {

namespace {

template <typename T>
inline __device__ T gpu_atomic_add(const T val, T* address);

template <>
inline __device__
float gpu_atomic_add(const float val, float* address) {
  return atomicAdd(address, val);
}

template <typename T>
__global__ void ReduceWithAttentionRegionForward(const int nthreads, 
                              const T* bottom_data, const T* attention_data,
                              const int num_inputs,
                              const int A, const int X,
                              const int iter, T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int idx = index / X;
    const int a = idx % A;
    const int n = idx / A;

    const int target_index = (n * num_inputs + iter) * A + a;
    top_data[index] += bottom_data[index] * attention_data[target_index];
  }
}

template <typename T>
__global__ void ReduceWithAttentionRegionBackward(const int nthreads, const T* input_grad,
                            const T* bottom_data, const int num_inputs,
                            const int A, const int X,
                            const int iter, T* output_grad) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int idx = index / X;
    const int a = idx % A;
    const int n = idx / A;

    const int target_index = (n * num_inputs + iter) * A + a;
    if (X == 1)
      output_grad[target_index] = input_grad[index] * bottom_data[index];
    else
      gpu_atomic_add(input_grad[index] * bottom_data[index], 
                    output_grad + target_index);
  } // CUDA_1D_KERNEL_LOOP
} // ReduceWithAttentionRegionBackward


} // namespace

template<>
bool ReduceWithAttentionRegionOp<float, HIPContext>::RunOnDevice() {
  // first calculate the final channel size
  const int num_inputs = InputSize() - 1;
  DCHECK_EQ(num_inputs, iter_);
  auto& Attention = Input(0);
  auto* Y = Output(0); 

  const int N = Attention.dim32(0);
  const int C = Attention.dim32(1);
  const int A = C / iter_;
  DCHECK_EQ(C % iter_, 0);

  const int D = Input(1).dim32(1);
  const int X = D / A;
  DCHECK_EQ(D % A, 0);

  // resize as the first input, or any input afterwards
  Y->ResizeLike(Input(1));
  const int output_size = Y->size();
  math::Set<float, HIPContext>(
       output_size, 0.f, Y->mutable_data<float>(), &context_);

  for (int iter=0; iter<num_inputs; iter++) {
    auto& Xstar = Input(iter+1);
    ReduceWithAttentionRegionForward<float>
        <<<CAFFE_GET_BLOCKS(output_size),
           CAFFE_CUDA_NUM_THREADS, 0,
        context_.hip_stream()>>>(
            output_size,
            Xstar.data<float>(),
            Attention.data<float>(),
            num_inputs, A, X, iter,
            Y->mutable_data<float>()
        );
  }
  return true;
}

template<>
bool ReduceWithAttentionRegionGradientOp<float, HIPContext>::RunOnDevice() {
  const int num_inputs = InputSize() - 2;
  DCHECK_EQ(num_inputs, iter_);
  auto& dY = Input(0);
  auto& Attention = Input(1);

  const int N = Attention.dim32(0);
  const int C = Attention.dim32(1);
  const int A = C / iter_;

  const int D = Input(2).dim32(1);
  const int X = D / A;

  auto* dA = Output(0);
  dA->ResizeLike(Attention);
  const int output_size = dY.size();
  // Must zero-out dA before accumulating gradients
  math::Set<float, HIPContext>(
       dA->size(), 0.f, dA->mutable_data<float>(), &context_);

  for (int iter=0; iter<num_inputs; iter++) {
    auto& Xstar = Input(iter+2);
    ReduceWithAttentionRegionBackward<float>
        <<<CAFFE_GET_BLOCKS(output_size),
           CAFFE_CUDA_NUM_THREADS, 0,
        context_.hip_stream()>>>(
            output_size,
            dY.data<float>(),
            Xstar.data<float>(),
            num_inputs, A, X, iter,
            dA->mutable_data<float>()
        );
  }
  return true;
}


REGISTER_HIP_OPERATOR(ReduceWithAttentionRegion,
                       ReduceWithAttentionRegionOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(ReduceWithAttentionRegionGradient,
                       ReduceWithAttentionRegionGradientOp<float, HIPContext>);
} // namespace caffe2