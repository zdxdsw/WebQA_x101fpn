#include "hip/hip_runtime.h"
#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/math.h"
#include "mul_conv_gate_op.h"

namespace caffe2 {

namespace {

template <typename T>
__global__ void MulConvGateForward(const int nthreads, 
                                const T* input1, const T* input2, 
                                const int channels, const int pixels,
                                T* output) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int idx = index % pixels;
    const int b = (index / pixels) / channels;

    output[index] = input1[index] * input2[b * pixels + idx];
  }
}

template <typename T>
__global__ void MulConvGateBackward(const int nthreads, 
    const T* input_grad, const T* input1, const T* input2,
    const int channels, const int pixels, T* output_grad1, T* output_grad2) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int idx = index % pixels;
    const int x = (index / pixels) % channels;
    const int b = (index / pixels) / channels;
    const int index2 = b * pixels + idx;

    // first deal with input 1
    output_grad1[index] = input_grad[index] * input2[index2];
    if (x == 0) {
      // also update input 2
      int iidx = b * channels * pixels + idx;
      T grad = 0.;
      for (int i=0; i<channels; i++) {
        grad += input_grad[iidx] * input1[iidx];
        iidx += pixels;
      }

      output_grad2[index2] = grad;
    }
  }
}

} // namespace

template<>
bool MulConvGateOp<float, HIPContext>::RunOnDevice() {
  auto& X1 = Input(0); // Input data 1
  auto& X2 = Input(1); // Input data 2
  auto* Y = Output(0); // Output data, summation of the two

  const int N = X1.dim32(0);
  const int C = X1.dim32(1);
  const int H = X1.dim32(2);
  const int W = X1.dim32(3);

  const int A = X2.dim32(1);
  DCHECK_EQ(N, X2.dim32(0));
  DCHECK_EQ(C % A, 0);
  DCHECK_EQ(H, X2.dim32(2));
  DCHECK_EQ(W, X2.dim32(3));

  const int pixels = H * W;
  const int X = C / A;

  // N, C, H, W
  Y->Resize(N, C, H, W);
  const int output_size = Y->size();
  MulConvGateForward<float><<<CAFFE_GET_BLOCKS(output_size), CAFFE_CUDA_NUM_THREADS,
                          0, context_.hip_stream()>>>(
                          output_size, X1.data<float>(), X2.data<float>(), 
                          X, pixels, Y->mutable_data<float>());

  return true;
}


template<>
bool MulConvGateGradientOp<float, HIPContext>::RunOnDevice() {
  auto& dY  = Input(0);  // Gradient of the output data
  auto& X1  = Input(1);  // Input data 1
  auto& X2  = Input(2);  // Input data 2
  auto* dX1 = Output(0); // Gradient of the input data 1
  auto* dX2 = Output(1); // Gradient of the input data 2

  const int C = X1.dim32(1);
  const int H = X1.dim32(2);
  const int W = X1.dim32(3);

  const int A = X2.dim32(1);
  const int X = C / A;

  const int pixels = H * W;
  const int output_size = dY.size();

  dX1->ResizeLike(X1);
  dX2->ResizeLike(X2);

  MulConvGateBackward<float><<<CAFFE_GET_BLOCKS(output_size),
                           CAFFE_CUDA_NUM_THREADS,
                           0, context_.hip_stream()>>>(
                                 output_size, 
                                 dY.data<float>(),
                                 X1.data<float>(),
                                 X2.data<float>(),
                                 X,
                                 pixels,
                                 dX1->mutable_data<float>(),
                                 dX2->mutable_data<float>());
  return true;
}


REGISTER_HIP_OPERATOR(MulConvGate,
                       MulConvGateOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(MulConvGateGradient,
                       MulConvGateGradientOp<float, HIPContext>);
} // namespace caffe2