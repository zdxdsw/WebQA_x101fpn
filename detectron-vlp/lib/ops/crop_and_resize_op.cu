#include "hip/hip_runtime.h"
#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/math.h"
#include "crop_and_resize_op.h"

#include <stdio.h>

namespace caffe2 {

namespace {

template <typename T>
inline __device__ T gpu_atomic_add(const T val, T* address);

template <>
inline __device__
float gpu_atomic_add(const float val, float* address) {
  return atomicAdd(address, val);
}

template <typename T>
__global__ void CropAndResizeForward(const int nthreads, const T* bottom_data,
    const float spatial_scale, const int batch, const int channels,
    const int height, const int width,
    const int pooled_height, const int pooled_width,
    const float* bottom_rois, T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    const int x = index % pooled_width;
    const int y = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;

    const float* offset_bottom_rois = bottom_rois + n * 5;
    const int roi_batch_ind = static_cast<int>(offset_bottom_rois[0]);

    if (roi_batch_ind < 0 || roi_batch_ind >= batch) {
      top_data[index] = 0.;
      continue;
    }

    const float x1 = offset_bottom_rois[1];
    const float y1 = offset_bottom_rois[2];
    const float x2 = offset_bottom_rois[3];
    const float y2 = offset_bottom_rois[4];

    // the distance between two sampled points are n-1, rather than n
    const float in_y = (y1 + (y2 - y1) * y / (pooled_height - 1)) * spatial_scale;
    const float in_x = (x1 + (x2 - x1) * x / (pooled_width - 1)) * spatial_scale;

    const int top_y_index = floorf(in_y);
    const int bottom_y_index = ceilf(in_y);
    const float y_lerp = in_y - top_y_index;

    const int left_x_index = floorf(in_x);
    const int right_x_index = ceilf(in_x);
    const float x_lerp = in_x - left_x_index;

    const int base_idx = c * height * width;
    const int top_offset = top_y_index * width;
    const int bottom_offset = bottom_y_index * width;

    // also includes the case of extrapolation
    float top_left, top_right, bottom_left, bottom_right;
    if (top_y_index >= 0 && top_y_index < height) {
      if (left_x_index >= 0 && left_x_index < width) 
        top_left = bottom_data[base_idx + top_offset + left_x_index];
      else
        top_left = 0.;
      if (right_x_index >= 0 && right_x_index < width)
        top_right = bottom_data[base_idx + top_offset + right_x_index];
      else
        top_right = 0.;
    } else {
      top_left = 0.;
      top_right = 0.;
    }

    if (bottom_y_index >= 0 && bottom_y_index < height) {
      if (left_x_index >= 0 && left_x_index < width)
        bottom_left = bottom_data[base_idx + bottom_offset + left_x_index];
      else
        bottom_left = 0.;
      if (right_x_index >= 0 && right_x_index < width)
        bottom_right = bottom_data[base_idx + bottom_offset + right_x_index];
      else
        bottom_right = 0.;
    } else {
      bottom_left = 0.;
      bottom_right = 0.;
    }

    const float top = top_left + (top_right - top_left) * x_lerp;
    const float bottom = bottom_left + (bottom_right - bottom_left) * x_lerp;

    top_data[index] = top + (bottom - top) * y_lerp;
  }
}

template <typename T>
__global__ void CropAndResizeBackward(const int nthreads, const T* input_grad,
    const int num_rois, const float spatial_scale, const int batch,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width,
    T* output_grad,
    const float* bottom_rois) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int x = index % pooled_width;
    int y = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const float* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = static_cast<int>(offset_bottom_rois[0]);

    if (roi_batch_ind < 0 || roi_batch_ind >= batch) {
      continue;
    }

    const float x1 = offset_bottom_rois[1];
    const float y1 = offset_bottom_rois[2];
    const float x2 = offset_bottom_rois[3];
    const float y2 = offset_bottom_rois[4];

    const float original_y = (y1 + (y2 - y1) * y / (pooled_height - 1)) * spatial_scale;
    const float original_x = (x1 + (x2 - x1) * x / (pooled_width - 1)) * spatial_scale;

    const int top_y_index = floorf(original_y);
    const int bottom_y_index = ceilf(original_y);
    const float y_lerp = original_y - top_y_index;

    const int left_x_index = floorf(original_x);
    const int right_x_index = ceilf(original_x);
    const float x_lerp = original_x - left_x_index;

    const int base_idx = c * height * width;
    const int top_offset = top_y_index * width;
    const int bottom_offset = bottom_y_index * width;

    if (top_y_index >= 0 && top_y_index < height) {
      const float dtop = (1 - y_lerp) * input_grad[index];
      if (left_x_index >= 0 && left_x_index < width)
        gpu_atomic_add(static_cast<T>((1 - x_lerp) * dtop), 
                      output_grad + (base_idx + top_offset + left_x_index));
      if (right_x_index >= 0 && right_x_index < width)
        gpu_atomic_add(static_cast<T>(x_lerp * dtop),
                      output_grad + (base_idx + top_offset + right_x_index));
    }

    if (bottom_y_index >= 0 && bottom_y_index < height) {
      const float dbottom = y_lerp * input_grad[index];
      if (left_x_index >= 0 && left_x_index < width) 
        gpu_atomic_add(static_cast<T>((1 - x_lerp) * dbottom), 
                      output_grad + (base_idx + bottom_offset + left_x_index));
      if (right_x_index >= 0 && right_x_index < width)
        gpu_atomic_add(static_cast<T>(x_lerp * dbottom), 
                      output_grad + (base_idx + bottom_offset + right_x_index));
    }
  } // CUDA_1D_KERNEL_LOOP
} // CropAndResizeBackward


} // namespace

template<>
bool CropAndResizeOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);  // Input data to pool
  auto& R = Input(1);  // RoIs
  auto* Y = Output(0); // RoI pooled data

  if (R.size() == 0) {
    // Handle empty rois
    Y->Resize(0, X.dim32(1), pooled_height_, pooled_width_);
    // The following mutable_data calls are needed to allocate the tensors
    Y->mutable_data<float>();
    return true;
  }

  Y->Resize(R.dim32(0), X.dim32(1), pooled_height_, pooled_width_);
  int output_size = Y->size();
  CropAndResizeForward<float>
      <<<CAFFE_GET_BLOCKS(output_size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          spatial_scale_,
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          pooled_height_,
          pooled_width_,
          R.data<float>(),
          Y->mutable_data<float>());
  return true;
}

template<>
bool CropAndResizeGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X  = Input(0);  // Input data to pool
  auto& R  = Input(1);  // RoIs
  auto& dY = Input(2);  // Gradient of net w.r.t. output of "forward" op
                        // (aka "gradOutput")
  auto* dX = Output(0); // Gradient of net w.r.t. input to "forward" op
                        // (aka "gradInput")

  dX->ResizeLike(X);

  // Must zero-out dX before accumulating gradients
  math::Set<float, HIPContext>(
      dX->size(), 0.f, dX->mutable_data<float>(), &context_);

  if (dY.size() > 0) {  // Handle possibly empty gradient if there were no rois
    CropAndResizeBackward<float>
        <<<CAFFE_GET_BLOCKS(dY.size()),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context_.hip_stream()>>>(
            dY.size(),
            dY.data<float>(),
            R.dim32(0),
            spatial_scale_,
            X.dim32(0),
            X.dim32(1),
            X.dim32(2),
            X.dim32(3),
            pooled_height_,
            pooled_width_,
            dX->mutable_data<float>(),
            R.data<float>());
  }
  return true;
}


REGISTER_HIP_OPERATOR(CropAndResize,
                       CropAndResizeOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(CropAndResizeGradient,
                       CropAndResizeGradientOp<float, HIPContext>);
} // namespace caffe2