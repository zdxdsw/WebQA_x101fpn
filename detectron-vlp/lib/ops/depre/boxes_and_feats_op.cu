#include "hip/hip_runtime.h"
#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/math.h"
#include "boxes_and_feats_op.h"

namespace caffe2 {

namespace {

template <typename T>
inline __device__ T gpu_atomic_add(const T val, T* address);

template <>
inline __device__
int gpu_atomic_add(const int val, int* address) {
  return atomicAdd(address, val);
}

inline __device__ float _clip_min(const float f, const float m) {
    return (f > m) ? (f) : (m);
}

inline __device__ float _clip_max(const float f, const float m) {
    return (f < m) ? (f) : (m);
}

template <typename T>
__global__ void BoxesAndFeatsForward(const int nthreads,
                                    const T* cls_pred,
                                    const T* box_pred,
                                    const T* anchor,
                                    const TIndex* YI,
                                    const T* YV, 
                                    const T* info,
                                    const T stride,
                                    const int A,
                                    const int num_classes,
                                    const int height,
                                    const int width,
                                    const int pixel,
                                    T* boxes_pointer,
                                    T* feats_pointer,
                                    int* stats_pointer) {
  CUDA_1D_KERNEL_LOOP(r, nthreads) {
    const T cls_prob_val = -YV[r];
    const int index = static_cast<int>(YI[r]);
    int ind = index;
    const int w = ind % width;
    ind /= width;
    const int h = ind % height;
    ind /= height;
    const int c = ind % num_classes;
    const int a = ind / num_classes;
    const int ap = a * 4;
    const int bp = (a * 4 * height + h) * width + w;

    const T x = w * stride;
    const T y = h * stride;

    const T x1 = x + anchor[ap];
    const T y1 = y + anchor[ap+1];
    const T x2 = x + anchor[ap+2];
    const T y2 = y + anchor[ap+3];

    const T dx = box_pred[bp];
    const T dy = box_pred[bp + pixel];
    T dw = box_pred[bp + 2 * pixel];
    T dh = box_pred[bp + 3 * pixel];
    dw = _clip_max(dw, 4.1351666);
    dh = _clip_max(dh, 4.1351666);

    // do box transform
    const T ww = x2 - x1 + 1.;
    const T hh = y2 - y1 + 1.;
    const T ctr_x = x1 + 0.5 * ww;
    const T ctr_y = y1 + 0.5 * hh;

    const T pred_ctr_x = dx * ww + ctr_x;
    const T pred_ctr_y = dy * hh + ctr_y;
    T pred_w = exp(dw) * ww;
    T pred_h = exp(dh) * hh;

    const T height_max = info[0] - 1.;
    const T width_max = info[1] - 1.;

    const T xx1 = _clip_max(_clip_min(pred_ctr_x - 0.5 * pred_w, 0.), width_max);
    const T yy1 = _clip_max(_clip_min(pred_ctr_y - 0.5 * pred_h, 0.), height_max);
    const T xx2 = _clip_max(_clip_min(pred_ctr_x + 0.5 * pred_w - 1., 0.), width_max);
    const T yy2 = _clip_max(_clip_min(pred_ctr_y + 0.5 * pred_h - 1., 0.), height_max); 

    pred_w = xx2 - xx1 + 1.;
    pred_h = yy2 - yy1 + 1.;

    // then dump the data
    const int bbp = r * 7;
    boxes_pointer[bbp] = xx1;
    boxes_pointer[bbp + 1] = yy1;
    boxes_pointer[bbp + 2] = xx2;
    boxes_pointer[bbp + 3] = yy2;
    // because of the bug, we need to first negate the cls_prob
    boxes_pointer[bbp + 4] = cls_prob_val;
    boxes_pointer[bbp + 5] = static_cast<float>(c);
    // valid: 1., invalid: 0.
    const T valid = (pred_w < 1. || pred_h < 1.) ? 0. : 1.;
    boxes_pointer[bbp + 6] = valid;

    // accumulate stats
    if (valid) {
      gpu_atomic_add(1, stats_pointer + c);
    }

    const int fp = r * num_classes;
    const int ffp = (a * num_classes * height + h) * width + w;
    // for loop to do the copy
    for (int j=0, jj=0; j<num_classes; j++, jj+=pixel) {
      feats_pointer[fp + j] = cls_pred[ffp + jj];
    }
  }
}

} // namespace

template<>
bool BoxesAndFeatsOp<float, HIPContext>::RunOnDevice() {
  auto& cls_preds = Input(0); 
  auto& box_preds = Input(1);
  auto& anchors = Input(2);
  auto& YI = Input(3);
  auto& YV = Input(4);
  auto& im_info = Input(5);

  // get some sizes
  const int N = cls_preds.dim32(0);
  const int num_cls = cls_preds.dim32(1) / A_;
  DCHECK_EQ(box_preds.dim32(1) / A_, 4);
  const int H = cls_preds.dim32(2);
  const int W = cls_preds.dim32(3);
  const int pixel = H * W;
  const int num_probs = A_ * num_cls * pixel;
  const int offset_cls = im_ * num_probs;
  const int offset_box = im_ * A_ * 4 * pixel;
  const int offset_info = im_ * 3;
  const float stride = pow(2., lvl_);
  const int R = YI.dim32(0);

  auto* boxes = Output(0);
  boxes->Resize(R, 7);
  auto* feats = Output(1);
  feats->Resize(R, num_cls);
  auto* stats = Output(2);
  stats->Resize(num_cls);
  int* stats_pointer = stats->mutable_data<int>();
  math::Set<int, HIPContext>(num_cls, 0, stats_pointer, &context_);

  BoxesAndFeatsForward<float><<<CAFFE_GET_BLOCKS(R), CAFFE_CUDA_NUM_THREADS,
                    0, context_.hip_stream()>>>(R, 
                                                 cls_preds.data<float>() + offset_cls,
                                                 box_preds.data<float>() + offset_box,
                                                 anchors.data<float>(),
                                                 YI.data<TIndex>(),
                                                 YV.data<float>(),
                                                 im_info.data<float>() + offset_info,
                                                 stride, A_, num_cls, H, W, pixel,
                                                 boxes->mutable_data<float>(),
                                                 feats->mutable_data<float>(),
                                                 stats_pointer);

  return true;
}

REGISTER_HIP_OPERATOR(BoxesAndFeats,
                       BoxesAndFeatsOp<float, HIPContext>);
} // namespace caffe2